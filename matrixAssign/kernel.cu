#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
static void runTest(void);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
	runTest();

    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.

	checkCudaErrors(hipDeviceReset());
 
    return 0;
}

__global__ void testKernel(float* g_idata, float* g_odata)
{
	extern __shared__ float sdata[]; // the 3rd parameter of the host kernel func. 
	const unsigned int bid = blockIdx.x;
	const unsigned int tid_in_block = threadIdx.x;
	const unsigned int tid_in_grid = blockDim.x * blockIdx.x + threadIdx.x;

#if 1	
	sdata[tid_in_block] = g_idata[tid_in_grid];
	__syncthreads();

	//sdata[tid_in_block] *= (float)bid;
	//sdata[tid_in_block] *= (float)tid_in_block;
	sdata[tid_in_block] *= (float)tid_in_grid;

	__syncthreads();
	g_odata[tid_in_grid] = sdata[tid_in_block];
#else
	__syncthreads();
	g_idata[tid_in_grid] *= (float)bid;
#endif
}


static void runTest()
{
	// set GPU device
	checkCudaErrors(hipSetDevice(0));

	unsigned int num_blocks = 4;
	unsigned int num_threads = 4;
	unsigned int mem_size = sizeof(float) * num_threads * num_blocks;

	// allocate host arrays
	float* h_idata = (float*)malloc(mem_size);
	assert(h_idata != NULL);
	float* h_odata = (float*)malloc(mem_size);
	assert(h_odata != NULL);

	// alllocate GPU arrays
	float* d_idata = NULL;
	checkCudaErrors(hipMalloc((void**)&d_idata, mem_size));
	float* d_odata = NULL;
	checkCudaErrors(hipMalloc((void**)&d_odata, mem_size));

	// initialize the host array (input)
	for (unsigned int i = 0; i < num_threads * num_blocks; i++)
		h_idata[i] = 1.0f;
	hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice);


	// The model will be like this:
	// 
	//  Grid#0
	//	Blk#0	Blk#1	Blk#2	Blk#3
	//  T0..T3  T4..T7  T8..T11 T12..T15
	//  
	// define the CUDA runtime parameters
	dim3 grid(num_blocks, 1, 1);
	dim3 threads(num_threads, 1, 1);

	// run the kernel
	testKernel <<<grid, threads, mem_size >>>(d_idata, d_odata);
	checkCudaErrors(hipDeviceSynchronize());
	hipMemcpy(h_odata, d_odata, mem_size, hipMemcpyDeviceToHost);

	// output 
	for (unsigned int i = 0; i < num_blocks; i++) {
		for (unsigned int j = 0; j < num_threads; j++) {
			printf("%5.0f", h_odata[i * num_threads + j]);
		}
		printf("\n");
	}

	// release host arrays memory
	free(h_idata);
	free(h_odata);

	checkCudaErrors(hipFree(d_idata));
	checkCudaErrors(hipFree(d_odata));
	checkCudaErrors(hipDeviceReset());
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
